#include "hip/hip_runtime.h"
//
// Created by nick on 4/3/18.
//
#include "hip/hip_runtime.h"
#include "parallel.h"
#include <stdio.h>

#define TILE_DIM 16

__global__ void coalescedMultiply(float *a, float *b, float *c, int N)
{
    __shared__ float aTile[TILE_DIM][TILE_DIM], bTile[TILE_DIM][TILE_DIM];

    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    float sum = 0.0f;
    aTile[threadIdx.y][threadIdx.x] = a[row*TILE_DIM+threadIdx.x];
    bTile[threadIdx.y][threadIdx.x] = b[threadIdx.y*N+col];
    __syncthreads();
    for (int i = 0; i < TILE_DIM; i++)
    {
        sum += aTile[threadIdx.y][i] * bTile[i][threadIdx.x];
    }
    c[row*N+col] = sum;
}


float rothVerf_parallel(int N)
{
    return 0.0f;
}

int main()
{
    printf("Hello!\n");
    return 0;
}