#include "hip/hip_runtime.h"
//
// Created by nick on 4/3/18.
//
#include "hip/hip_runtime.h"
#include "parallel.h"
#include <stdio.h>
#include <host_defines.h>
#include <>
#include <hip/hip_runtime.h>
#include <sys/time.h>

#define TILE_DIM 32

__global__ void coalescedMultiply(float *a, float *b, float *c, int N)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    float sum = 0.0f;
    for (int i = 0; i < TILE_DIM; i++) {
        sum += a[row*TILE_DIM+i] * b[i*N+col];
    }
    c[row*N+col] = sum;
    /*__shared__ float aTile[TILE_DIM][TILE_DIM], bTile[TILE_DIM][TILE_DIM];

    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    float sum = 0.0f;
    aTile[threadIdx.y][threadIdx.x] = a[row*TILE_DIM+threadIdx.x];
    bTile[threadIdx.y][threadIdx.x] = b[threadIdx.y*N+col];
    __syncthreads();
    for (int i = 0; i < TILE_DIM; i++)
    {
        sum += aTile[threadIdx.y][i] * bTile[i][threadIdx.x];
    }
    c[row*N+col] = sum;*/
}

static void HandleError( hipError_t err,
                         const char *file,
                         int line ) {
    if (err != hipSuccess) {
        printf( "%s in %s at line %d\n", hipGetErrorString( err ),
                file, line );
        exit( EXIT_FAILURE );
    }
}
#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))

// gets the current time in seconds with microsecond precision
double get_time()
{
    struct timeval t;
    struct timezone tzp;
    gettimeofday(&t, &tzp);
    return t.tv_sec + t.tv_usec * 1e-6;
}

void fillIdentity(int N, float *matrix, MAT_POS pos, float scalar)
{
    int i_start, j_start, i_end, j_end;
    findStart(N, pos, &i_start, &j_start, &i_end, &j_end);
    for (int i = i_start; i < i_end; i++)
    {
        for (int j = j_start; j < j_end; j++)
        {
            matrix[i*N+j] = (i == j) ? (scalar) : 0.0f;
        }
    }
}

void fillZeros(int N, float *matrix, MAT_POS pos)
{
    int i_start, j_start, i_end, j_end;
    findStart(N, pos, &i_start, &j_start, &i_end, &j_end);
    for (int i = i_start; i < i_end; i++)
    {
        for (int j = j_start; j < j_end; j++)
        {
            matrix[i*N+j] = 0.0f;
        }
    }
}

void fillRand(int N, float *matrix, MAT_POS pos, float scalar)
{
    int i_start, j_start, i_end, j_end;
    findStart(N, pos, &i_start, &j_start, &i_end, &j_end);
    for (int i = i_start; i < i_end; i++)
    {
        for (int j = j_start; j < j_end; j++)
        {
            matrix[i*N+j] = scalar * (rand() / (float)RAND_MAX);
        }
    }
}

#define fabs(val) (val) < 0.0f ? (-(val)) : (val)

float mat_diff(int N, float *matrix1, float *matrix2)
{
    float diff = 0.0f;
    for (int ij = 0; ij < (N*N); ij++)
    {
        int i = ij / N;
        int j = (ij / N) % N;
        diff += fabs(matrix1[i*N+j] - matrix2[i*N+j]);
    }
    return diff;
}

void printMa(int N, float *matrix)
{
    for (int i = 0; i < N; i++)
    {
        for (int j = 0; j < N; j++)
        {
            printf("%.1f ", matrix[i*N+j]);
        }
        printf("\n");
    }
}

float rothVerf_parallel(int ext)
{
    int N = ext*2;
    // Configuration
    dim3 threadsPerBlock(TILE_DIM, TILE_DIM);
    dim3 numBlocks(N / threadsPerBlock.x, N / threadsPerBlock.y);
    srand(100);

    // Memory allocation
    float *matrix_1, *matrix_2, *result;
    matrix_1 = (float*)malloc(N*N*sizeof(float));
    matrix_2 = (float*)malloc(N*N*sizeof(float));
    result = (float*)malloc(N*N*sizeof(float));

    float *d_m1, *d_m2, *d_m3;
    HANDLE_ERROR(hipMalloc((void**)&d_m1, N*N*sizeof(float)));
    HANDLE_ERROR(hipMalloc((void**)&d_m2, N*N*sizeof(float)));
    HANDLE_ERROR(hipMalloc((void**)&d_m3, N*N*sizeof(float)));

    // As per serial, fill these matrices with the right values
    fillIdentity(N, matrix_1, MAT_TL, 1.0f);
    fillRand(N, matrix_1, MAT_TR, 1.0f);
    fillZeros(N, matrix_1, MAT_BL);
    fillIdentity(N, matrix_1, MAT_BR, 1.0f);

    fillIdentity(N, matrix_2, MAT_TL, 1.0f);
    fillRand(N, matrix_2, MAT_TR, 2.0f);
    fillZeros(N, matrix_2, MAT_BL);
    fillIdentity(N, matrix_2, MAT_BR, -1.0f);


    // Copy over the first two matrices, and set up our result matrix
    HANDLE_ERROR(hipMemcpy(d_m1, matrix_1, N*N*sizeof(float), hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(d_m2, matrix_2, N*N*sizeof(float), hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemset(d_m3, 0, N*N*sizeof(float)));

    // Do the multiplication
    coalescedMultiply<<<numBlocks, threadsPerBlock>>>(d_m1, d_m2, d_m3, N);

    HANDLE_ERROR(hipDeviceSynchronize()); // Wait for completion

    // So now d_m3 is our result matrix
    // Re-use matrix_1 for last matrix, to memcpy over to dm_1
    fillIdentity(N, matrix_1, MAT_TL, 1.0f);
    fillRand(N, matrix_1, MAT_TR, -1.0f);
    fillZeros(N, matrix_1, MAT_BL);
    fillIdentity(N, matrix_1, MAT_BR, 1.0f);

    // Multiply d_m3 * d_m1 into d_m2
    // But first set our result matrix to all 0s
    HANDLE_ERROR(hipMemset(d_m2, 0, N*N*sizeof(float)));
    // Now copy over the new matrix_1
    HANDLE_ERROR(hipMemcpy(d_m1, matrix_1, N*N*sizeof(float), hipMemcpyHostToDevice));
    coalescedMultiply<<<numBlocks, threadsPerBlock>>>(d_m3, d_m1, d_m2, N);

    HANDLE_ERROR(hipDeviceSynchronize()); // Wait for completion

    // Copy over d_m2 into matrix_1 for comparison
    HANDLE_ERROR(hipMemcpy(matrix_1, d_m2, N*N*sizeof(float), hipMemcpyDeviceToHost));

    // Re-use result for RHS matrix
    fillIdentity(N, result, MAT_TL, 1.0f);
    fillZeros(N, result, MAT_TR);
    fillZeros(N, result, MAT_BL);
    fillIdentity(N, result, MAT_BR, -1.0f);

    printf("LHS:\n");
    printMa(N, matrix_1);

    // Get the error sum
    float error = mat_diff(N, result, matrix_1);

    // Free up memory
    free(matrix_1);
    free(matrix_2);
    free(result);

    HANDLE_ERROR(hipFree(d_m1));
    HANDLE_ERROR(hipFree(d_m2));
    HANDLE_ERROR(hipFree(d_m3));

    // Return the error sum
    return error;
}

int main()
{
    float err = rothVerf_parallel(10);
    printf("Error is %.1f\n", err);
    return 0;
}