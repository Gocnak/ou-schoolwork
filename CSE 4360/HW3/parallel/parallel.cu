#include "hip/hip_runtime.h"
//
// Created by nick on 4/3/18.
//
#include "hip/hip_runtime.h"
#include <stdio.h>
#include <host_defines.h>
#include <>
#include <hip/hip_runtime.h>
#include <sys/time.h>

#define TILE_DIM 16

typedef enum
{
    MAT_TL = 0, // Top left
    MAT_TR = 1, // Top right
    MAT_BL = 2, // Bottom left
    MAT_BR = 3 // Bottom right
} MAT_POS;


inline void findStart(int N, MAT_POS pos, int *i, int *j, int *i_end, int *j_end)
{
    int isTop = (pos < MAT_BL);
    int isRight = (pos % 2 == 1);

    *i = (isTop) ? 0 : N/2;
    *j = (isRight) ? N/2 : 0;
    *i_end = (isTop) ? N/2 : N;
    *j_end = (isRight) ? N : N/2;
}

__global__ void coalescedMultiply(float *a, float *b, float *c, int N)
{
    __shared__ float aTile[TILE_DIM][TILE_DIM], bTile[TILE_DIM][TILE_DIM];

    int row = blockIdx.y * TILE_DIM + threadIdx.y;
    int col = blockIdx.x * TILE_DIM + threadIdx.x;
    float sum = 0.0f;
    int idx;
    for (int sub = 0; sub < gridDim.x; ++sub)
    {
        idx = row * N + sub * TILE_DIM + threadIdx.x;
        if (idx >= N*N)
        {
            aTile[threadIdx.y][threadIdx.x] = 0;
        }
        else {
            aTile[threadIdx.y][threadIdx.x] = a[idx];
        }

        idx = (sub * TILE_DIM + threadIdx.y) * N + col;
        if (idx >= N*N)
        {
            bTile[threadIdx.y][threadIdx.x] = 0;
        }
        else {
            bTile[threadIdx.y][threadIdx.x] = b[idx];
        }

        __syncthreads();

        for (int k = 0; k < TILE_DIM; k++)
        {
            sum += aTile[threadIdx.y][k] * bTile[k][threadIdx.x];
        }

        __syncthreads();
    }

    if (row < N && col < N)
        c[row * N + col] = sum;
}

static void HandleError( hipError_t err,
                         const char *file,
                         int line ) {
    if (err != hipSuccess) {
        printf( "%s in %s at line %d\n", hipGetErrorString( err ),
                file, line );
        exit( EXIT_FAILURE );
    }
}
#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))

// gets the current time in seconds with microsecond precision
double get_time()
{
    struct timeval t;
    gettimeofday(&t, NULL);
    return t.tv_sec + t.tv_usec * 1e-6;
}

void fillIdentity(int N, float *matrix, MAT_POS pos, float scalar)
{
    int i_start, j_start, i_end, j_end;
    findStart(N, pos, &i_start, &j_start, &i_end, &j_end);
    for (int i = i_start; i < i_end; i++)
    {
        for (int j = j_start; j < j_end; j++)
        {
            matrix[i*N+j] = (i == j) ? (scalar) : 0.0f;
        }
    }
}

void fillZeros(int N, float *matrix, MAT_POS pos)
{
    int i_start, j_start, i_end, j_end;
    findStart(N, pos, &i_start, &j_start, &i_end, &j_end);
    for (int i = i_start; i < i_end; i++)
    {
        for (int j = j_start; j < j_end; j++)
        {
            matrix[i*N+j] = 0.0f;
        }
    }
}

void fillRand(int N, float *matrix, MAT_POS pos, float scalar)
{
    int i_start, j_start, i_end, j_end;
    findStart(N, pos, &i_start, &j_start, &i_end, &j_end);
    for (int i = i_start; i < i_end; i++)
    {
        for (int j = j_start; j < j_end; j++)
        {
            matrix[i*N+j] = scalar * (rand() / (float)RAND_MAX);
        }
    }
}

#define fabs(val) (val) < 0.0f ? (-(val)) : (val)

float mat_diff(int N, float *matrix1, float *matrix2)
{
    float diff = 0.0f;
    for (int ij = 0; ij < (N*N); ij++)
    {
        int i = ij / N;
        int j = (ij / N) % N;
        diff += fabs(matrix1[i*N+j] - matrix2[i*N+j]);
    }
    return diff;
}

void printMa(int N, float *matrix)
{
    for (int i = 0; i < N; i++)
    {
        for (int j = 0; j < N; j++)
        {
            printf("%.1f ", matrix[i*N+j]);
        }
        printf("\n");
    }
}

float rothVerf_parallel(int ext)
{
    int N = ext*2;
    // Configuration
    dim3 dimGrid(((N + TILE_DIM - 1) / TILE_DIM), ((N + TILE_DIM - 1) / TILE_DIM));
    dim3 dimBlock(TILE_DIM, TILE_DIM);
    srand(100);

    // Memory allocation
    float *matrix_1, *matrix_2, *result;
    matrix_1 = (float*)malloc(N*N*sizeof(float));
    matrix_2 = (float*)malloc(N*N*sizeof(float));
    result = (float*)malloc(N*N*sizeof(float));

    hipEvent_t start, stop;
    HANDLE_ERROR(hipEventCreate(&start));
    HANDLE_ERROR(hipEventCreate(&stop));

    HANDLE_ERROR(hipEventRecord(start, 0));

    float *d_m1, *d_m2, *d_m3;
    HANDLE_ERROR(hipMalloc((void**)&d_m1, N*N*sizeof(float)));
    HANDLE_ERROR(hipMalloc((void**)&d_m2, N*N*sizeof(float)));
    HANDLE_ERROR(hipMalloc((void**)&d_m3, N*N*sizeof(float)));

    // As per serial, fill these matrices with the right values
    fillIdentity(N, matrix_1, MAT_TL, 1.0f);
    fillRand(N, matrix_1, MAT_TR, 1.0f);
    fillZeros(N, matrix_1, MAT_BL);
    fillIdentity(N, matrix_1, MAT_BR, 1.0f);

    fillIdentity(N, matrix_2, MAT_TL, 1.0f);
    fillRand(N, matrix_2, MAT_TR, 2.0f);
    fillZeros(N, matrix_2, MAT_BL);
    fillIdentity(N, matrix_2, MAT_BR, -1.0f);


    // Copy over the first two matrices, and set up our result matrix
    HANDLE_ERROR(hipMemcpy(d_m1, matrix_1, N*N*sizeof(float), hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(d_m2, matrix_2, N*N*sizeof(float), hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemset(d_m3, 0, N*N*sizeof(float)));

    // Do the multiplication
    coalescedMultiply<<<dimGrid, dimBlock>>>(d_m1, d_m2, d_m3, N);

    HANDLE_ERROR(hipDeviceSynchronize()); // Wait for completion

    // So now d_m3 is our result matrix
    // Re-use matrix_1 for last matrix, to memcpy over to dm_1
    fillIdentity(N, matrix_1, MAT_TL, 1.0f);
    fillRand(N, matrix_1, MAT_TR, -1.0f);
    fillZeros(N, matrix_1, MAT_BL);
    fillIdentity(N, matrix_1, MAT_BR, 1.0f);

    // Multiply d_m3 * d_m1 into d_m2
    // But first set our result matrix to all 0s
    HANDLE_ERROR(hipMemset(d_m2, 0, N*N*sizeof(float)));
    // Now copy over the new matrix_1
    HANDLE_ERROR(hipMemcpy(d_m1, matrix_1, N*N*sizeof(float), hipMemcpyHostToDevice));
    coalescedMultiply<<<dimGrid, dimBlock>>>(d_m3, d_m1, d_m2, N);

    HANDLE_ERROR(hipDeviceSynchronize()); // Wait for completion

    // Copy over d_m2 into matrix_1 for comparison
    HANDLE_ERROR(hipMemcpy(matrix_1, d_m2, N*N*sizeof(float), hipMemcpyDeviceToHost));

    HANDLE_ERROR(hipEventRecord(stop, 0));
    HANDLE_ERROR(hipEventSynchronize(stop));

    float gpu_elapsed_ms;
    HANDLE_ERROR(hipEventElapsedTime(&gpu_elapsed_ms, start, stop));
    printf("Done calculating! Elapsed time: %.1f ms\n", gpu_elapsed_ms);

    // Re-use result for RHS matrix
    fillIdentity(N, result, MAT_TL, 1.0f);
    fillZeros(N, result, MAT_TR);
    fillZeros(N, result, MAT_BL);
    fillIdentity(N, result, MAT_BR, -1.0f);

    //printf("LHS:\n");
    //printMa(N, matrix_1);

    // Get the error sum
    float error = mat_diff(N, result, matrix_1);

    // Free up memory
    free(matrix_1);
    free(matrix_2);
    free(result);

    HANDLE_ERROR(hipFree(d_m1));
    HANDLE_ERROR(hipFree(d_m2));
    HANDLE_ERROR(hipFree(d_m3));

    // Return the error sum
    return error;
}

int main(int argc, char** argv)
{
    int N = 5000;
    if (argc > 1)
        N = atoi(argv[1]);

    printf("Calculating CUDA with N=%d\n", N);
    double start = get_time();
    float err = rothVerf_parallel(N);
    printf("Error is %.1f in total runtime of %.2f ms\n", err, (get_time() - start) * 1000.0f);
    return 0;
}