#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include <stdio.h>
#include <host_defines.h>
#include <>
#include <hip/hip_runtime.h>
#include <sys/time.h>
#include <hipblas.h>

#define TILE_DIM 16

typedef enum
{
    MAT_TL = 0, // Top left
    MAT_TR = 1, // Top right
    MAT_BL = 2, // Bottom left
    MAT_BR = 3 // Bottom right
} MAT_POS;


inline void findStart(int N, MAT_POS pos, int *i, int *j, int *i_end, int *j_end)
{
    int isTop = (pos < MAT_BL);
    int isRight = (pos % 2 == 1);

    *i = (isTop) ? 0 : N/2;
    *j = (isRight) ? N/2 : 0;
    *i_end = (isTop) ? N/2 : N;
    *j_end = (isRight) ? N : N/2;
}

static void HandleError( hipError_t err,
                         const char *file,
                         int line ) {
    if (err != hipSuccess) {
        printf( "%s in %s at line %d\n", hipGetErrorString( err ),
                file, line );
        exit( EXIT_FAILURE );
    }
}
#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))

// gets the current time in seconds with microsecond precision
double get_time()
{
    struct timeval t;
    struct timezone tzp;
    gettimeofday(&t, &tzp);
    return t.tv_sec + t.tv_usec * 1e-6;
}

void fillIdentity(int N, float *matrix, MAT_POS pos, float scalar)
{
    int i_start, j_start, i_end, j_end;
    findStart(N, pos, &i_start, &j_start, &i_end, &j_end);
    for (int i = i_start; i < i_end; i++)
    {
        for (int j = j_start; j < j_end; j++)
        {
            matrix[i*N+j] = (i == j) ? (scalar) : 0.0f;
        }
    }
}

void fillZeros(int N, float *matrix, MAT_POS pos)
{
    int i_start, j_start, i_end, j_end;
    findStart(N, pos, &i_start, &j_start, &i_end, &j_end);
    for (int i = i_start; i < i_end; i++)
    {
        for (int j = j_start; j < j_end; j++)
        {
            matrix[i*N+j] = 0.0f;
        }
    }
}

void fillRand(int N, float *matrix, MAT_POS pos, float scalar)
{
    int i_start, j_start, i_end, j_end;
    findStart(N, pos, &i_start, &j_start, &i_end, &j_end);
    for (int i = i_start; i < i_end; i++)
    {
        for (int j = j_start; j < j_end; j++)
        {
            matrix[i*N+j] = scalar * (rand() / (float)RAND_MAX);
        }
    }
}

#define fabs(val) (val) < 0.0f ? (-(val)) : (val)

float mat_diff(int N, float *matrix1, float *matrix2)
{
    float diff = 0.0f;
    for (int ij = 0; ij < (N*N); ij++)
    {
        int i = ij / N;
        int j = (ij / N) % N;
        diff += fabs(matrix1[i*N+j] - matrix2[i*N+j]);
    }
    return diff;
}

void printMa(int N, float *matrix)
{
    for (int i = 0; i < N; i++)
    {
        for (int j = 0; j < N; j++)
        {
            printf("%.1f ", matrix[i*N+j]);
        }
        printf("\n");
    }
}

float rothVerf_parallel(int ext)
{
    int N = ext*2;
    // Configuration
    dim3 dimGrid(((N + TILE_DIM - 1) / TILE_DIM), ((N + TILE_DIM - 1) / TILE_DIM));
    dim3 dimBlock(TILE_DIM, TILE_DIM);
    srand(100);

    // Memory allocation
    float *matrix_1, *matrix_2, *result;
    matrix_1 = (float*)malloc(N*N*sizeof(float));
    matrix_2 = (float*)malloc(N*N*sizeof(float));
    result = (float*)malloc(N*N*sizeof(float));

    hipEvent_t start, stop;
    HANDLE_ERROR(hipEventCreate(&start));
    HANDLE_ERROR(hipEventCreate(&stop));

    HANDLE_ERROR(hipEventRecord(start, 0));

    hipblasHandle_t handle;
    hipblasStatus_t stat = hipblasCreate(&handle);

    if (stat != HIPBLAS_STATUS_SUCCESS)
    {
        printf("!!!! Cublas init error!\n");
        return -1.0f;
    }


    float *d_m1, *d_m2, *d_m3;
    HANDLE_ERROR(hipMalloc((void**)&d_m1, N*N*sizeof(float)));
    HANDLE_ERROR(hipMalloc((void**)&d_m2, N*N*sizeof(float)));
    HANDLE_ERROR(hipMalloc((void**)&d_m3, N*N*sizeof(float)));

    // As per serial, fill these matrices with the right values
    fillIdentity(N, matrix_1, MAT_TL, 1.0f);
    fillRand(N, matrix_1, MAT_TR, 1.0f);
    fillZeros(N, matrix_1, MAT_BL);
    fillIdentity(N, matrix_1, MAT_BR, 1.0f);

    fillIdentity(N, matrix_2, MAT_TL, 1.0f);
    fillRand(N, matrix_2, MAT_TR, 2.0f);
    fillZeros(N, matrix_2, MAT_BL);
    fillIdentity(N, matrix_2, MAT_BR, -1.0f);


    // Copy over the first two matrices, and set up our result matrix
    HANDLE_ERROR(hipMemcpy(d_m1, matrix_1, N*N*sizeof(float), hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(d_m2, matrix_2, N*N*sizeof(float), hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemset(d_m3, 0, N*N*sizeof(float)));

    // Do the multiplication
    const float alpha = 1.0f;
    const float beta = 0.0f;
    stat = hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, N, N, &alpha, d_m1, N, d_m2, N, &beta, d_m3, N);
    if (stat != HIPBLAS_STATUS_SUCCESS)
    {
        printf("Failed the first multiply!\n");
        return -1.0f;
    }

    HANDLE_ERROR(hipDeviceSynchronize()); // Wait for completion

    // So now d_m3 is our result matrix
    // Re-use matrix_1 for last matrix, to memcpy over to dm_1
    fillIdentity(N, matrix_1, MAT_TL, 1.0f);
    fillRand(N, matrix_1, MAT_TR, -1.0f);
    fillZeros(N, matrix_1, MAT_BL);
    fillIdentity(N, matrix_1, MAT_BR, 1.0f);

    // Multiply d_m3 * d_m1 into d_m2
    // But first set our result matrix to all 0s
    HANDLE_ERROR(hipMemset(d_m2, 0, N*N*sizeof(float)));
    // Now copy over the new matrix_1
    HANDLE_ERROR(hipMemcpy(d_m1, matrix_1, N*N*sizeof(float), hipMemcpyHostToDevice));

    stat = hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, N, N, &alpha, d_m3, N, d_m1, N, &beta, d_m2, N);
    if (stat != HIPBLAS_STATUS_SUCCESS)
    {
        printf("Failed the second multiply!\n");
        return -1.0f;
    }

    HANDLE_ERROR(hipDeviceSynchronize()); // Wait for completion

    // Copy over d_m2 into matrix_1 for comparison
    HANDLE_ERROR(hipMemcpy(matrix_1, d_m2, N*N*sizeof(float), hipMemcpyDeviceToHost));

    HANDLE_ERROR(hipEventRecord(stop, 0));
    HANDLE_ERROR(hipEventSynchronize(stop));

    float gpu_elapsed_ms;
    HANDLE_ERROR(hipEventElapsedTime(&gpu_elapsed_ms, start, stop));
    printf("Done calculating! Elapsed time: %.1f ms\n", gpu_elapsed_ms);

    // Re-use result for RHS matrix
    fillIdentity(N, result, MAT_TL, 1.0f);
    fillZeros(N, result, MAT_TR);
    fillZeros(N, result, MAT_BL);
    fillIdentity(N, result, MAT_BR, -1.0f);

    //printf("LHS:\n");
    //printMa(N, matrix_1);

    // Get the error sum
    float error = mat_diff(N, result, matrix_1);

    // Free up memory
    free(matrix_1);
    free(matrix_2);
    free(result);

    HANDLE_ERROR(hipFree(d_m1));
    HANDLE_ERROR(hipFree(d_m2));
    HANDLE_ERROR(hipFree(d_m3));

    // Return the error sum
    return error;
}

int main(int argc, char** argv)
{
    int N = 5000;
    if (argc > 1)
        N = atoi(argv[1]);

    printf("Calculating cudaBLAS with N=%d\n", N);
    double start = get_time();
    float err = rothVerf_parallel(N);
    printf("Error is %.1f, total time in %.2f ms", err, (get_time() - start) * 1000.0f);
    return 0;
}